#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define CHECK_FOR_ERRORS(err) if (err != hipSuccess) {\
                                printf("%s:\n%s in %s in line %\n",hipGetErrorName(err), hipGetErrorString(err), __FILE__, __LINE__);\
                                exit(EXIT_FAILURE);\
                              }

__global__ void addKernel(float *c, const float*a, const float*b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int sizeBytes = arraySize * sizeof(float);
    const float a[arraySize] = { 1, 2, 3, 4, 5 };
    const float b[arraySize] = { 10, 20, 30, 40, 50 };
    float c[arraySize] = { 0 };

    float * dev_a = 0;
    float * dev_b = 0;
    float * dev_c = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    CHECK_FOR_ERRORS(cudaStatus);

    // allocating memory
    cudaStatus = hipMalloc((void**)&dev_a, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);

    cudaStatus = hipMalloc((void**)&dev_b, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);

    cudaStatus = hipMalloc((void**)&dev_c, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);

    // init with 0
    
    cudaStatus = hipMemset((void*)dev_a, 0, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);

    cudaStatus = hipMemset((void*)dev_b, 0, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);

    cudaStatus = hipMemset((void*)dev_c, 0, sizeBytes);
    CHECK_FOR_ERRORS(cudaStatus);
    

    //copy to memory
    cudaStatus = hipMemcpy(dev_a, a, sizeBytes, hipMemcpyHostToDevice);
    CHECK_FOR_ERRORS(cudaStatus);

    cudaStatus = hipMemcpy(dev_b, b, sizeBytes, hipMemcpyHostToDevice);
    CHECK_FOR_ERRORS(cudaStatus);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, arraySize >> > (dev_c, dev_a, dev_b);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    CHECK_FOR_ERRORS(cudaStatus);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    CHECK_FOR_ERRORS(cudaStatus);
    
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, sizeBytes, hipMemcpyDeviceToHost);
    CHECK_FOR_ERRORS(cudaStatus);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    CHECK_FOR_ERRORS(cudaStatus);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
